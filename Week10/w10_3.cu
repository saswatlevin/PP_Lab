
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void rev(char *a,int *len)
{
	int id=threadIdx.x;
	int val=a[id];
	int k=1;
	int sum=0;
	while(val>0)
	{
		int rem=val%2;
		rem*=k;
		k*=10;
		val/=2;
		sum+=rem;
	}
	printf("%c\t%d\n",a[id],sum);
}

int main()
{
	char a[20];
	int *d_m;
	char *d_a;

	printf("Enter String:");
	scanf("%s",a);

	int size = sizeof(int)*strlen(a);
	int len=strlen(a);

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_m,sizeof(int));

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_m, &len, sizeof(int), hipMemcpyHostToDevice);

	rev<<<1,len>>>(d_a,d_m);
	hipFree(d_a);
	hipFree(d_m);
}
