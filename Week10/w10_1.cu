#include "hip/hip_runtime.h"
include <stdio.h>

__global__ void sub(char *a,char *b,int *lenA,int *lenB)
{
	int id=threadIdx.x;
	int flag=1;
	for(int i=0;i<*lenB;i++)
	{
		if(a[id+i]!=b[i])
		{
			flag=0;
			break;
		}
	}
	if(flag==1)
		printf("FOUND\n");
}

int main()
{
	char a[20], b[20];
	int *d_m,*d_v;
	char *d_a, *d_b;

	printf("Enter String:");
	scanf("%s",a);
	printf("Enter Sub String:");
	scanf("%s",b);

	int sizeA = sizeof(char)*strlen(a);
	int sizeB = sizeof(char)*strlen(b);
	int lenA=strlen(a);
	int lenB=strlen(b);

	hipMalloc((void**)&d_a,sizeA);
	hipMalloc((void**)&d_b,sizeB);
	hipMalloc((void**)&d_m,sizeof(int));
	hipMalloc((void**)&d_v,sizeof(int));

	hipMemcpy(d_a, &a, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeB, hipMemcpyHostToDevice);
	hipMemcpy(d_m, &lenA, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_v, &lenB, sizeof(int), hipMemcpyHostToDevice);

	sub<<<1,lenA>>>(d_a, d_b,d_m,d_v);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_m);
	hipFree(d_v);
}

