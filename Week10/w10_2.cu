
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void rev(char *a,char *b,int *len)
{
	int id=threadIdx.x;
	b[id]=a[*len-id-1];
	b[*len-id-1]=a[id];
}

int main(void)
{
	char a[20], b[20];
	int *d_m;
	char *d_a, *d_b;

	printf("Enter String:");
	scanf("%s",a);

	int size = sizeof(int)*strlen(a);
	int len=strlen(a);

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_m,sizeof(int));

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_m, &len, sizeof(int), hipMemcpyHostToDevice);

	rev<<<1,len>>>(d_a, d_b,d_m);

	hipMemcpy(&b,d_b, size, hipMemcpyDeviceToHost);
	printf("%s\n",b);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_m);
}
